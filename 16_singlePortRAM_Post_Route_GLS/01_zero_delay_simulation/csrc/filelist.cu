LDVERSION= $(shell $(PIC_LD) -v | grep -q 2.30 ;echo $$?)
ifeq ($(LDVERSION), 0)
     LD_NORELAX_FLAG= --no-relax
endif

ARCHIVE_OBJS=
ARCHIVE_OBJS += _1681597_archive_1.so
_1681597_archive_1.so : archive.0/_1681597_archive_1.a
	@$(AR) -s $<
	@$(PIC_LD) -shared  -Bsymbolic $(LD_NORELAX_FLAG)  -o .//../test.daidir//_1681597_archive_1.so --whole-archive $< --no-whole-archive
	@rm -f $@
	@ln -sf .//../test.daidir//_1681597_archive_1.so $@




VCS_CU_ARC_OBJS = 


O0_OBJS =

$(O0_OBJS) : %.o: %.c
	$(CC_CG) $(CFLAGS_O0) -c -o $@ $<


%.o: %.c
	$(CC_CG) $(CFLAGS_CG) -c -o $@ $<
CU_UDP_OBJS = \
objs/udps/iJuDZ.o objs/udps/Nginy.o objs/udps/QRIPd.o objs/udps/U9Crq.o objs/udps/eLUxc.o  \
objs/udps/Tcvek.o objs/udps/i0psV.o objs/udps/TB6Ix.o objs/udps/aLBRD.o objs/udps/V7WvH.o  \
objs/udps/FqLFq.o objs/udps/yz1uR.o objs/udps/RUKVA.o objs/udps/afYMY.o objs/udps/fsCp2.o  \
objs/udps/USMy8.o objs/udps/PSwnp.o objs/udps/f0xYg.o objs/udps/tw5vQ.o objs/udps/qwM0m.o  \
objs/udps/KME4Y.o objs/udps/h5bqa.o objs/udps/Gdmn6.o objs/udps/U6qjI.o objs/udps/DtgyT.o  \
objs/udps/K79QG.o objs/udps/dUm5G.o objs/udps/ymc5r.o objs/udps/AENcr.o objs/udps/wIL7Z.o  \
objs/udps/uQKHy.o objs/udps/dxA5k.o 

CU_LVL_OBJS = \
SIM_l.o 

MAIN_OBJS = \
objs/amcQw_d.o 

CU_OBJS = $(MAIN_OBJS) $(ARCHIVE_OBJS) $(CU_UDP_OBJS) $(CU_LVL_OBJS)

